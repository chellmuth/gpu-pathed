#include "render_session.h"

#include "camera.h"

#define checkCudaErrors(result) { gpuAssert((result), __FILE__, __LINE__); }
static void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace rays {

const Vec3 defaultAlbedo = Vec3(0.45098f, 0.823529f, 0.0862745f);

RenderSession::RenderSession()
{
    m_pathTracer = std::make_unique<PathTracer>();

    m_cudaGlobals = std::make_unique<CUDAGlobals>();
    m_scene = std::make_unique<Scene>(defaultAlbedo);
    m_sceneModel = std::make_unique<SceneModel>(
        m_pathTracer.get(),
        m_scene.get(),
        defaultAlbedo,
        defaultLightPosition
    );
    m_sceneModel->subscribe([this]() {
        m_pathTracer->reset();
        m_scene->setColor(m_sceneModel->getMaterialIndex(), m_sceneModel->getColor());

        checkCudaErrors(hipMemcpy(
            dev_materials,
            m_scene->getMaterialsData(),
            m_scene->getMaterialsSize(),
            hipMemcpyHostToDevice
        ));

        createWorld<<<1, 1>>>(
            dev_primitives,
            dev_materials,
            m_cudaGlobals->d_world,
            m_sceneModel->getLightPosition(),
            true
        );

        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
    });
}

void RenderSession::init(
    GLuint pbo,
    int width,
    int height
) {
    m_width = width;
    m_height = height;

    const Camera camera(
        Vec3(0.f, 0.3f, 5.f),
        30.f / 180.f * M_PI,
        { width, height }
    );
    m_cudaGlobals->copyCamera(camera);

    checkCudaErrors(hipMalloc((void **)&dev_primitives, primitiveCount * sizeof(Primitive *)));
    checkCudaErrors(hipMalloc((void **)&dev_materials, materialCount * sizeof(Material)));
    m_cudaGlobals->mallocWorld();

    m_scene->init();
    checkCudaErrors(hipMemcpy(
        dev_materials,
        m_scene->getMaterialsData(),
        m_scene->getMaterialsSize(),
        hipMemcpyHostToDevice
    ));

    createWorld<<<1, 1>>>(
        dev_primitives,
        dev_materials,
        m_cudaGlobals->d_world,
        m_sceneModel->getLightPosition(),
        false
    );

    checkCudaErrors(hipGetLastError());

    m_pathTracer->init(pbo, width, height);
}

SceneModel& RenderSession::getSceneModel()
{
    return *m_sceneModel;
}


}

#undef checkCudaErrors
