#include "hip/hip_runtime.h"
#include "camera.h"

#include <cmath>

#include "vec3.h"

namespace rays {

__device__ Camera::Camera(const Vec3 &origin, float verticalFOV, const Resolution &resolution)
    : m_origin(origin),
      m_verticalFOV(verticalFOV),
      m_resolution(resolution)
{}

__device__ Ray Camera::generateRay(int row, int col, hiprandState &randState) const
{
    const float top = std::tan(m_verticalFOV / 2.f);
    const float height = top * 2.f;

    const float aspectRatio = 1.f * m_resolution.x / m_resolution.y;
    const float width = height * aspectRatio;
    const float right = width / 2.f;

    const float xCanonical = (col + hiprand_uniform(&randState)) / m_resolution.x;
    const float yCanonical = (row + hiprand_uniform(&randState)) / m_resolution.y;

    const float y = yCanonical * height - top;
    const float x = xCanonical * width - right;

    const Vec3 direction = normalized(Vec3(x, y, -1));

    return Ray(m_origin, direction);
}

}
