#include "hip/hip_runtime.h"
#include "camera.h"

#include <cmath>

namespace rays {

Camera::Camera(
    const Vec3 &origin,
    const Vec3 &target,
    const Vec3 &up,
    float verticalFOV,
    const Resolution &resolution
) : m_origin(origin),
    m_target(target),
    m_up(up),
    m_verticalFOV(verticalFOV),
    m_resolution(resolution)
{
    m_cameraToWorld = lookAt(m_origin, m_target, m_up);
}

__device__ Ray Camera::generateRay(int row, int col) const
{
    return generateRay(row, col, make_float2(0.5f, 0.5f));
}

__device__ Ray Camera::generateRay(int row, int col, hiprandState &randState) const
{
    const float xi1 = hiprand_uniform(&randState);
    const float xi2 = hiprand_uniform(&randState);
    return generateRay(row, col, make_float2(xi1, xi2));
}

__device__ Ray Camera::generateRay(int row, int col, float2 samples) const
{
    const float top = std::tan(m_verticalFOV / 2.f);
    const float height = top * 2.f;

    const float aspectRatio = 1.f * m_resolution.x / m_resolution.y;
    const float width = height * aspectRatio;
    const float right = width / 2.f;

    const float xCanonical = (col + samples.x) / m_resolution.x;
    const float yCanonical = (row + samples.y) / m_resolution.y;

    const float y = yCanonical * height - top;
    const float x = xCanonical * width - right;

    const Vec3 direction = normalized(Vec3(x, y, -1));

    const Vec3 origin(0.f);
    const Ray transformedRay = m_cameraToWorld.apply(Ray(origin, direction));
    return transformedRay;

    return Ray(m_origin, direction);
}

}
