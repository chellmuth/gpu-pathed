#include "hip/hip_runtime.h"
#include "path_tracer.h"

#include <iostream>

#include <cfloat>
#include <cuda_gl_interop.h>

#include "camera.h"
#include "frame.h"
#include "primitive.h"
#include "material.h"
#include "scene.h"
#include "vec3.h"

#define checkCudaErrors(result) { gpuAssert((result), __FILE__, __LINE__); }
static void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace rays {

static constexpr bool debug = false;
static const Vec3 defaultAlbedo = Vec3(0.45098f, 0.823529f, 0.0862745f);
static constexpr float defaultLightPosition = -0.6f;

PathTracer::PathTracer()
    : m_currentSamples(0)
{
    m_sceneModel = std::make_unique<SceneModel>(defaultAlbedo, defaultLightPosition);
    m_sceneModel->subscribe([this]() {
        m_currentSamples = 0;

        rays::createWorld<<<1, 1>>>(
            dev_primitives,
            dev_world,
            m_sceneModel->getColor(),
            m_sceneModel->getLightPosition(),
            true
        );

        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
    });
}

__global__ static void renderInit(int width, int height, hiprandState *randState)
{
    const int row = threadIdx.y + blockIdx.y * blockDim.y;
    const int col = threadIdx.x + blockIdx.x * blockDim.x;

    if ((row >= height) || (col >= width)) return;
    const int pixelIndex = row * width + col;

    constexpr int seed = 0;
    hiprand_init(seed, pixelIndex, 0, &randState[pixelIndex]);
}

__device__ static Vec3 calculateLi(const Ray& ray, PrimitiveList **world, hiprandState &randState)
{
    Vec3 beta = Vec3(1.f);
    Vec3 result = Vec3(0.f);

    HitRecord record;
    bool hit = (*world)->hit(ray, 0.f, FLT_MAX, record);
    if (hit) {
        const Vec3 emit = record.materialPtr->emit(record);

        if (!emit.isZero()) {
            result += emit * beta;
        }
    } else {
        const Vec3 direction = normalized(ray.direction());
        const float t = 0.5f * (direction.y() + 1.0f);
        return Vec3(1.f - t) + t * Vec3(0.5f, 0.7f, 1.f);
    }

    for (int path = 2; path < 10; path++) {
        const Frame intersection(record.normal);
        float pdf;

        const Vec3 wi = record.materialPtr->sample(record, &pdf, randState);
        const Vec3 bounceDirection = intersection.toWorld(wi);

        beta *= record.materialPtr->f(record.wo, wi) * intersection.cosTheta(wi) / pdf;

        const Ray bounceRay(record.point, bounceDirection);
        hit = (*world)->hit(bounceRay, 1e-3, FLT_MAX, record);
        if (hit) {
            const Vec3 emit = record.materialPtr->emit(record);
            if (!emit.isZero()) {
                result += emit * beta;
            }
        } else {
            const Vec3 direction = normalized(ray.direction());
            const float t = 0.5f * (direction.y() + 1.f);
            const Vec3 skyRadiance = (Vec3(1.f - t) + t * Vec3(0.5f, 0.7f, 1.f)) * 0.5f;
            return result + skyRadiance * beta;
        }
    }

    return result;
}

__global__ static void renderKernel(
    uchar4 *fb,
    Vec3 *radiances,
    int spp,
    int currentSamples,
    int width, int height,
    PrimitiveList **world,
    hiprandState *randState
) {
    const int row = threadIdx.y + blockIdx.y * blockDim.y;
    const int col = threadIdx.x + blockIdx.x * blockDim.x;
    if ((row >= height) || (col >= width)) { return; }

    const int pixelIndex = row * width + col;

    hiprandState &localRand = randState[pixelIndex];
    const Camera camera(
        Vec3(0.f, 0.3f, 5.f),
        30.f / 180.f * M_PI,
        { width, height }
    );

    for (int sample = 1; sample <= spp; sample++) {
        const Ray cameraRay = camera.generateRay(row, col, localRand);
        const Vec3 Li = calculateLi(cameraRay, world, localRand);

        const int spp = currentSamples + sample;

        Vec3 next;
        if (spp > 1) {
            const Vec3 current = radiances[pixelIndex];
            next = current * (spp - 1) / spp + (Li / spp);
        } else {
            next = Li;
        }

        radiances[pixelIndex] = next;
    }

    const Vec3 finalRadiance = radiances[pixelIndex];

    fb[pixelIndex].x = fmaxf(0.f, fminf(1.f, finalRadiance.x())) * 255;
    fb[pixelIndex].y = fmaxf(0.f, fminf(1.f, finalRadiance.y())) * 255;
    fb[pixelIndex].z = fmaxf(0.f, fminf(1.f, finalRadiance.z())) * 255;
    fb[pixelIndex].w = 255;
}

void PathTracer::init(
    GLuint pbo,
    int width,
    int height
) {
    checkCudaErrors(
        hipGraphicsGLRegisterBuffer(
            &m_cudaPbo,
            pbo,
            hipGraphicsRegisterFlagsWriteDiscard
        )
    );

    m_width = width;
    m_height = height;
    const int pixelCount = m_width * m_height;

    checkCudaErrors(hipMalloc((void **)&dev_randState, pixelCount * sizeof(hiprandState)));
    checkCudaErrors(hipMalloc((void **)&dev_primitives, rays::primitiveCount * sizeof(Primitive *)));
    checkCudaErrors(hipMalloc((void **)&dev_world, sizeof(PrimitiveList *)));
    checkCudaErrors(hipMalloc((void **)&dev_radiances, pixelCount * sizeof(Vec3)));

    rays::createWorld<<<1, 1>>>(
        dev_primitives,
        dev_world,
        m_sceneModel->getColor(),
        m_sceneModel->getLightPosition(),
        false
    );

    checkCudaErrors(hipGetLastError());

    dim3 blocks(m_width, m_height);
    renderInit<<<blocks, 1>>>(m_width, m_height, dev_randState);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

void PathTracer::render()
{
    checkCudaErrors(hipGraphicsMapResources(1, &m_cudaPbo, NULL));
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dev_map, NULL, m_cudaPbo));

    const int samplesPerPass = 8;

    const int blockWidth = 16;
    const int blockHeight = 16;

    const dim3 blocks(m_width / blockWidth + 1, m_height / blockHeight + 1);
    const dim3 threads(blockWidth, blockHeight);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    renderKernel<<<blocks, threads>>>(
        dev_map,
        dev_radiances,
        samplesPerPass,
        m_currentSamples,
        m_width, m_height,
        dev_world,
        dev_randState
    );

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    if (debug) {
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "CUDA Frame: " << milliseconds << std::endl;
    }

    m_currentSamples += samplesPerPass;
    m_sceneModel->updateSpp(m_currentSamples);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGraphicsUnmapResources(1, &m_cudaPbo, NULL));
}

SceneModel& PathTracer::getSceneModel()
{
    return *m_sceneModel;
}

}
