#include "hip/hip_runtime.h"
#include "cuda_globals.h"

#include <iostream>

#include "scene.h"

#define checkCudaErrors(result) { gpuAssert((result), __FILE__, __LINE__); }
static void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace rays {

void CUDAGlobals::copyCamera(const Camera &camera)
{
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera)));

    checkCudaErrors(hipMemcpy(
        d_camera,
        &camera,
        sizeof(Camera),
        hipMemcpyHostToDevice
    ));
}

__global__ static void initWorldKernel(
    PrimitiveList *world,
    Triangle *triangles,
    Sphere *spheres,
    Material *materials
) {
    if (blockIdx.x != 0 || blockIdx.y != 0) { return; }
    if (threadIdx.x != 0 || threadIdx.y != 0) { return; }

    *world = PrimitiveList(
        triangles,
        triangleCount,
        spheres,
        sphereCount,
        materials,
        materialCount
    );
}

void CUDAGlobals::mallocWorld()
{
    checkCudaErrors(hipMalloc((void **)&d_materials, materialCount * sizeof(Material)));

    checkCudaErrors(hipMalloc((void **)&d_triangles, triangleCount * sizeof(Triangle)));
    checkCudaErrors(hipMalloc((void **)&d_spheres, sphereCount * sizeof(Sphere)));

    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(PrimitiveList)));

    initWorldKernel<<<1, 1>>>(d_world, d_triangles, d_spheres, d_materials);
    checkCudaErrors(hipDeviceSynchronize());

}


}

#undef checkCudaErrors
