#include "hip/hip_runtime.h"
#include "cuda_globals.h"

#include "macro_helper.h"
#include "scene.h"

#define checkCudaErrors(result) { gpuAssert((result), __FILE__, __LINE__); }

namespace rays {

void CUDAGlobals::mallocCamera()
{
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera)));
}

void CUDAGlobals::copyCamera(const Camera &camera)
{
    checkCudaErrors(hipMemcpy(
        d_camera,
        &camera,
        sizeof(Camera),
        hipMemcpyHostToDevice
    ));
}

__global__ static void updateMaterialLookup(
    MaterialLookup *materialLookup,
    MaterialIndex *materialIndices,
    Lambertian *lambertians,
    Mirror *mirrors,
    Glass *glasses
) {
    if (blockIdx.x != 0 || blockIdx.y != 0) { return; }
    if (threadIdx.x != 0 || threadIdx.y != 0) { return; }

    materialLookup->indices = materialIndices;
    materialLookup->lambertians = lambertians;
    materialLookup->mirrors = mirrors;
    materialLookup->glasses = glasses;
}

__global__ static void initWorldKernel(
    PrimitiveList *world,
    Triangle *triangles,
    int triangleSize,
    Sphere *spheres,
    int sphereSize,
    int *lightIndices,
    int lightIndexSize,
    MaterialLookup *materialLookup
) {
    if (blockIdx.x != 0 || blockIdx.y != 0) { return; }
    if (threadIdx.x != 0 || threadIdx.y != 0) { return; }

    *world = PrimitiveList(
        triangles,
        triangleSize,
        spheres,
        sphereSize,
        lightIndices,
        lightIndexSize,
        materialLookup
    );
}

void CUDAGlobals::mallocMaterials(const SceneData &sceneData)
{
    const int lambertianSize = sceneData.materialStore.getLambertians().size();
    const int mirrorSize = sceneData.materialStore.getMirrors().size();
    const int glassSize = sceneData.materialStore.getGlasses().size();

    const std::vector<MaterialIndex> &indices = sceneData.materialStore.getIndices();

    checkCudaErrors(hipMalloc((void **)&d_materialIndices, indices.size() * sizeof(MaterialIndex)));
    checkCudaErrors(hipMalloc((void **)&d_lambertians, lambertianSize * sizeof(Lambertian)));
    checkCudaErrors(hipMalloc((void **)&d_mirrors, mirrorSize * sizeof(Mirror)));
    checkCudaErrors(hipMalloc((void **)&d_glasses, glassSize * sizeof(Glass)));
}

void CUDAGlobals::copyMaterials(const SceneData &sceneData)
{
    const std::vector<MaterialIndex> indices = sceneData.materialStore.getIndices();
    checkCudaErrors(hipMemcpy(
        d_materialIndices,
        indices.data(),
        indices.size() * sizeof(MaterialIndex),
        hipMemcpyHostToDevice
    ));

    const std::vector<Lambertian> &lambertians = sceneData.materialStore.getLambertians();
    checkCudaErrors(hipMemcpy(
        d_lambertians,
        lambertians.data(),
        lambertians.size() * sizeof(Lambertian),
        hipMemcpyHostToDevice
    ));

    const std::vector<Mirror> &mirrors = sceneData.materialStore.getMirrors();
    checkCudaErrors(hipMemcpy(
        d_mirrors,
        mirrors.data(),
        mirrors.size() * sizeof(Mirror),
        hipMemcpyHostToDevice
    ));

    const std::vector<Glass> &glasses = sceneData.materialStore.getGlasses();
    checkCudaErrors(hipMemcpy(
        d_glasses,
        glasses.data(),
        glasses.size() * sizeof(Glass),
        hipMemcpyHostToDevice
    ));

    updateMaterialLookup<<<1, 1>>>(
        d_materialLookup,
        d_materialIndices,
        d_lambertians,
        d_mirrors,
        d_glasses
    );

    checkCudaErrors(hipDeviceSynchronize());
}

void CUDAGlobals::freeMaterials()
{
    checkCudaErrors(hipFree(d_materialIndices));
    checkCudaErrors(hipFree(d_lambertians));
    checkCudaErrors(hipFree(d_mirrors));
    checkCudaErrors(hipFree(d_glasses));
}

void CUDAGlobals::mallocWorld(const SceneData &sceneData)
{
    const int triangleSize = sceneData.triangles.size();
    const int sphereSize = sceneData.spheres.size();
    const int lightIndexSize = sceneData.lightIndices.size();

    checkCudaErrors(hipMalloc((void **)&d_triangles, triangleSize * sizeof(Triangle)));
    checkCudaErrors(hipMalloc((void **)&d_spheres, sphereSize * sizeof(Sphere)));
    checkCudaErrors(hipMalloc((void **)&d_lightIndices, lightIndexSize * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_materialLookup, sizeof(MaterialLookup)));

    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(PrimitiveList)));

    initWorldKernel<<<1, 1>>>(
        d_world,
        d_triangles,
        triangleSize,
        d_spheres,
        sphereSize,
        d_lightIndices,
        lightIndexSize,
        d_materialLookup
    );
    checkCudaErrors(hipDeviceSynchronize());
}

void CUDAGlobals::copySceneData(const SceneData &sceneData)
{
    checkCudaErrors(hipMemcpy(
        d_triangles,
        sceneData.triangles.data(),
        sceneData.triangles.size() * sizeof(Triangle),
        hipMemcpyHostToDevice
    ));

    checkCudaErrors(hipMemcpy(
        d_spheres,
        sceneData.spheres.data(),
        sceneData.spheres.size() * sizeof(Sphere),
        hipMemcpyHostToDevice
    ));

    checkCudaErrors(hipMemcpy(
        d_lightIndices,
        sceneData.lightIndices.data(),
        sceneData.lightIndices.size() * sizeof(int),
        hipMemcpyHostToDevice
    ));
}

}
