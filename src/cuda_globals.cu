#include "hip/hip_runtime.h"
#include "cuda_globals.h"

#include <iostream>

#include "scene.h"

#define checkCudaErrors(result) { gpuAssert((result), __FILE__, __LINE__); }
static void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace rays {

void CUDAGlobals::copyCamera(const Camera &camera)
{
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera)));

    checkCudaErrors(hipMemcpy(
        d_camera,
        &camera,
        sizeof(Camera),
        hipMemcpyHostToDevice
    ));
}

__global__ static void initWorldKernel(
    PrimitiveList *world,
    Triangle *triangles,
    int triangleSize,
    Sphere *spheres,
    int sphereSize,
    Material *materials,
    int materialSize
) {
    if (blockIdx.x != 0 || blockIdx.y != 0) { return; }
    if (threadIdx.x != 0 || threadIdx.y != 0) { return; }

    *world = PrimitiveList(
        triangles,
        triangleSize,
        spheres,
        sphereSize,
        materials,
        materialSize
    );
}

void CUDAGlobals::mallocWorld(const SceneData &sceneData)
{
    const int materialSize = sceneData.materials.size();
    const int triangleSize = sceneData.triangles.size();
    const int sphereSize = sceneData.spheres.size();

    checkCudaErrors(hipMalloc((void **)&d_materials, materialSize * sizeof(Material)));

    checkCudaErrors(hipMalloc((void **)&d_triangles, triangleSize * sizeof(Triangle)));
    checkCudaErrors(hipMalloc((void **)&d_spheres, sphereSize * sizeof(Sphere)));

    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(PrimitiveList)));

    initWorldKernel<<<1, 1>>>(
        d_world,
        d_triangles,
        triangleSize,
        d_spheres,
        sphereSize,
        d_materials,
        materialSize
    );
    checkCudaErrors(hipDeviceSynchronize());
}

void CUDAGlobals::copySceneData(const SceneData &sceneData)
{
    checkCudaErrors(hipMemcpy(
        d_triangles,
        sceneData.triangles.data(),
        sceneData.triangles.size() * sizeof(Triangle),
        hipMemcpyHostToDevice
    ));

    checkCudaErrors(hipMemcpy(
        d_spheres,
        sceneData.spheres.data(),
        sceneData.spheres.size() * sizeof(Sphere),
        hipMemcpyHostToDevice
    ));
}

}

#undef checkCudaErrors
