#include "cuda_globals.h"

#include <iostream>

#define checkCudaErrors(result) { gpuAssert((result), __FILE__, __LINE__); }
static void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace rays {

void CUDAGlobals::copyCamera(const Camera &camera)
{
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera)));

    checkCudaErrors(hipMemcpy(
        d_camera,
        &camera,
        sizeof(Camera),
        hipMemcpyHostToDevice
    ));
}

}

#undef checkCudaErrors
