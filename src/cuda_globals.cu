#include "hip/hip_runtime.h"
#include "cuda_globals.h"

#include "macro_helper.h"
#include "scene.h"

#define checkCudaErrors(result) { gpuAssert((result), __FILE__, __LINE__); }

namespace rays {

void CUDAGlobals::mallocCamera()
{
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera)));
}

void CUDAGlobals::copyCamera(const Camera &camera)
{
    checkCudaErrors(hipMemcpy(
        d_camera,
        &camera,
        sizeof(Camera),
        hipMemcpyHostToDevice
    ));
}

__global__ static void initWorldKernel(
    World *world,
    Triangle *triangles,
    int triangleSize,
    Sphere *spheres,
    int sphereSize,
    int *lightIndices,
    int lightIndexSize,
    EnvironmentLight *environmentLight,
    MaterialLookup *materialLookup
) {
    if (blockIdx.x != 0 || blockIdx.y != 0) { return; }
    if (threadIdx.x != 0 || threadIdx.y != 0) { return; }

    *world = World(
        triangles,
        triangleSize,
        spheres,
        sphereSize,
        lightIndices,
        lightIndexSize,
        environmentLight,
        materialLookup
    );
}

void CUDAGlobals::initMaterials(const SceneData &sceneData)
{
    checkCudaErrors(hipMalloc((void **)&d_materialLookup, sizeof(MaterialLookup)));

    m_materialLookup.mallocMaterials(sceneData);
    m_materialLookup.copyMaterials(sceneData);

    checkCudaErrors(hipMemcpy(
        d_materialLookup,
        &m_materialLookup,
        sizeof(MaterialLookup),
        hipMemcpyHostToDevice
    ));
}

void CUDAGlobals::updateMaterials(const SceneData &sceneData)
{
    m_materialLookup.freeMaterials();
    m_materialLookup.mallocMaterials(sceneData);
    m_materialLookup.copyMaterials(sceneData);

    checkCudaErrors(hipMemcpy(
        d_materialLookup,
        &m_materialLookup,
        sizeof(MaterialLookup),
        hipMemcpyHostToDevice
    ));
}

void CUDAGlobals::mallocWorld(const SceneData &sceneData)
{
    const int triangleSize = sceneData.triangles.size();
    const int sphereSize = sceneData.spheres.size();
    const int lightIndexSize = sceneData.lightIndices.size();

    checkCudaErrors(hipMalloc((void **)&d_triangles, triangleSize * sizeof(Triangle)));
    checkCudaErrors(hipMalloc((void **)&d_spheres, sphereSize * sizeof(Sphere)));
    checkCudaErrors(hipMalloc((void **)&d_lightIndices, lightIndexSize * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_environmentLight, sizeof(EnvironmentLight)));

    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(World)));

    initWorldKernel<<<1, 1>>>(
        d_world,
        d_triangles,
        triangleSize,
        d_spheres,
        sphereSize,
        d_lightIndices,
        lightIndexSize,
        d_environmentLight,
        d_materialLookup
    );
    checkCudaErrors(hipDeviceSynchronize());

    const auto &environmentLightParams = sceneData.environmentLightParams;
    m_environmentLight = environmentLightParams.createEnvironmentLight();
}

void CUDAGlobals::copySceneData(const SceneData &sceneData)
{
    checkCudaErrors(hipMemcpy(
        d_triangles,
        sceneData.triangles.data(),
        sceneData.triangles.size() * sizeof(Triangle),
        hipMemcpyHostToDevice
    ));

    checkCudaErrors(hipMemcpy(
        d_spheres,
        sceneData.spheres.data(),
        sceneData.spheres.size() * sizeof(Sphere),
        hipMemcpyHostToDevice
    ));

    checkCudaErrors(hipMemcpy(
        d_lightIndices,
        sceneData.lightIndices.data(),
        sceneData.lightIndices.size() * sizeof(int),
        hipMemcpyHostToDevice
    ));

    checkCudaErrors(hipMemcpy(
        d_environmentLight,
        &m_environmentLight,
        sizeof(EnvironmentLight),
        hipMemcpyHostToDevice
    ));
}

}
