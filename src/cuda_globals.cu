#include "hip/hip_runtime.h"
#include "cuda_globals.h"

#include "macro_helper.h"
#include "scene.h"

#define checkCudaErrors(result) { gpuAssert((result), __FILE__, __LINE__); }

namespace rays {

void CUDAGlobals::mallocCamera()
{
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera)));
}

void CUDAGlobals::copyCamera(const Camera &camera)
{
    checkCudaErrors(hipMemcpy(
        d_camera,
        &camera,
        sizeof(Camera),
        hipMemcpyHostToDevice
    ));
}

__global__ static void initWorldKernel(
    PrimitiveList *world,
    Triangle *triangles,
    int triangleSize,
    Sphere *spheres,
    int sphereSize,
    int *lightIndices,
    int lightIndexSize,
    Material *lambertians,
    int lambertianSize,
    MaterialLookup *materialLookup
) {
    if (blockIdx.x != 0 || blockIdx.y != 0) { return; }
    if (threadIdx.x != 0 || threadIdx.y != 0) { return; }

    materialLookup->lambertians = lambertians;
    materialLookup->lambertianSize = lambertianSize;

    *world = PrimitiveList(
        triangles,
        triangleSize,
        spheres,
        sphereSize,
        lightIndices,
        lightIndexSize,
        materialLookup
    );
}

void CUDAGlobals::mallocWorld(const SceneData &sceneData)
{
    const int lambertianSize = sceneData.lambertians.size();
    const int triangleSize = sceneData.triangles.size();
    const int sphereSize = sceneData.spheres.size();
    const int lightIndexSize = sceneData.lightIndices.size();

    checkCudaErrors(hipMalloc((void **)&d_lambertians, lambertianSize * sizeof(Material)));
    checkCudaErrors(hipMalloc((void **)&d_dummies, 1 * sizeof(Dummy)));

    checkCudaErrors(hipMalloc((void **)&d_materialLookup, sizeof(MaterialLookup)));

    checkCudaErrors(hipMalloc((void **)&d_triangles, triangleSize * sizeof(Triangle)));
    checkCudaErrors(hipMalloc((void **)&d_spheres, sphereSize * sizeof(Sphere)));
    checkCudaErrors(hipMalloc((void **)&d_lightIndices, lightIndexSize * sizeof(int)));

    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(PrimitiveList)));

    initWorldKernel<<<1, 1>>>(
        d_world,
        d_triangles,
        triangleSize,
        d_spheres,
        sphereSize,
        d_lightIndices,
        lightIndexSize,
        d_lambertians,
        lambertianSize,
        d_materialLookup
    );
    checkCudaErrors(hipDeviceSynchronize());
}

void CUDAGlobals::copySceneData(const SceneData &sceneData)
{
    checkCudaErrors(hipMemcpy(
        d_triangles,
        sceneData.triangles.data(),
        sceneData.triangles.size() * sizeof(Triangle),
        hipMemcpyHostToDevice
    ));

    checkCudaErrors(hipMemcpy(
        d_spheres,
        sceneData.spheres.data(),
        sceneData.spheres.size() * sizeof(Sphere),
        hipMemcpyHostToDevice
    ));

    checkCudaErrors(hipMemcpy(
        d_lightIndices,
        sceneData.lightIndices.data(),
        sceneData.lightIndices.size() * sizeof(int),
        hipMemcpyHostToDevice
    ));
}

}
