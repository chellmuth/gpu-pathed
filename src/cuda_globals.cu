#include "hip/hip_runtime.h"
#include "cuda_globals.h"

#include "macro_helper.h"
#include "scene.h"

#define checkCudaErrors(result) { gpuAssert((result), __FILE__, __LINE__); }

namespace rays {

void CUDAGlobals::mallocCamera()
{
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera)));
}

void CUDAGlobals::copyCamera(const Camera &camera)
{
    checkCudaErrors(hipMemcpy(
        d_camera,
        &camera,
        sizeof(Camera),
        hipMemcpyHostToDevice
    ));
}

__global__ static void updateMaterialLookup(
    MaterialLookup *materialLookup,
    MaterialIndex *materialIndices,
    Lambertian *lambertians,
    Mirror *mirrors,
    Glass *glasses
) {
    if (blockIdx.x != 0 || blockIdx.y != 0) { return; }
    if (threadIdx.x != 0 || threadIdx.y != 0) { return; }

    materialLookup->indices = materialIndices;
    materialLookup->lambertians = lambertians;
    materialLookup->mirrors = mirrors;
    materialLookup->glasses = glasses;
}

__global__ static void initWorldKernel(
    PrimitiveList *world,
    Triangle *triangles,
    int triangleSize,
    Sphere *spheres,
    int sphereSize,
    int *lightIndices,
    int lightIndexSize,
    MaterialLookup *materialLookup
) {
    if (blockIdx.x != 0 || blockIdx.y != 0) { return; }
    if (threadIdx.x != 0 || threadIdx.y != 0) { return; }

    *world = PrimitiveList(
        triangles,
        triangleSize,
        spheres,
        sphereSize,
        lightIndices,
        lightIndexSize,
        materialLookup
    );
}

void CUDAGlobals::mallocMaterials(const SceneData &sceneData)
{
    int lambertianSize = 0;
    int mirrorSize = 0;
    int glassSize = 0;

    for (const auto &param : sceneData.materialParams) {
        switch (param->getMaterialType()) {
        case MaterialType::Lambertian: {
            lambertianSize += 1;
            break;
        }
        case MaterialType::Mirror: {
            mirrorSize += 1;
            break;
        }
        case MaterialType::Glass: {
            glassSize += 1;
            break;
        }
        }
    }

    checkCudaErrors(hipMalloc((void **)&d_materialIndices, sceneData.materialParams.size() * sizeof(MaterialIndex)));
    checkCudaErrors(hipMalloc((void **)&d_lambertians, lambertianSize * sizeof(Lambertian)));
    checkCudaErrors(hipMalloc((void **)&d_mirrors, mirrorSize * sizeof(Mirror)));
    checkCudaErrors(hipMalloc((void **)&d_glasses, glassSize * sizeof(Glass)));
}

void CUDAGlobals::copyMaterials(const SceneData &sceneData)
{
    std::vector<MaterialIndex> indices;
    std::vector<Lambertian> lambertians;
    std::vector<Mirror> mirrors;
    std::vector<Glass> glasses;

    for (const auto &param : sceneData.materialParams) {
        switch (param->getMaterialType()) {
        case MaterialType::Lambertian: {
            Lambertian lambertian(*param);
            lambertians.push_back(lambertian);

            indices.push_back({MaterialType::Lambertian, lambertians.size() - 1});
            break;
        }
        case MaterialType::Mirror: {
            Mirror mirror(*param);
            mirrors.push_back(mirror);

            indices.push_back({MaterialType::Mirror, mirrors.size() - 1});
            break;
        }
        case MaterialType::Glass: {
            Glass glass(*param);
            glasses.push_back(glass);

            indices.push_back({MaterialType::Glass, glasses.size() - 1});
            break;
        }
        }
    }

    checkCudaErrors(hipMemcpy(
        d_materialIndices,
        indices.data(),
        indices.size() * sizeof(MaterialIndex),
        hipMemcpyHostToDevice
    ));

    checkCudaErrors(hipMemcpy(
        d_lambertians,
        lambertians.data(),
        lambertians.size() * sizeof(Lambertian),
        hipMemcpyHostToDevice
    ));

    checkCudaErrors(hipMemcpy(
        d_mirrors,
        mirrors.data(),
        mirrors.size() * sizeof(Mirror),
        hipMemcpyHostToDevice
    ));

    checkCudaErrors(hipMemcpy(
        d_glasses,
        glasses.data(),
        glasses.size() * sizeof(Glass),
        hipMemcpyHostToDevice
    ));

    updateMaterialLookup<<<1, 1>>>(
        d_materialLookup,
        d_materialIndices,
        d_lambertians,
        d_mirrors,
        d_glasses
    );

    checkCudaErrors(hipDeviceSynchronize());
}

void CUDAGlobals::freeMaterials()
{
    checkCudaErrors(hipFree(d_materialIndices));
    checkCudaErrors(hipFree(d_lambertians));
    checkCudaErrors(hipFree(d_mirrors));
    checkCudaErrors(hipFree(d_glasses));
}

void CUDAGlobals::mallocWorld(const SceneData &sceneData)
{
    const int triangleSize = sceneData.triangles.size();
    const int sphereSize = sceneData.spheres.size();
    const int lightIndexSize = sceneData.lightIndices.size();

    checkCudaErrors(hipMalloc((void **)&d_triangles, triangleSize * sizeof(Triangle)));
    checkCudaErrors(hipMalloc((void **)&d_spheres, sphereSize * sizeof(Sphere)));
    checkCudaErrors(hipMalloc((void **)&d_lightIndices, lightIndexSize * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_materialLookup, sizeof(MaterialLookup)));

    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(PrimitiveList)));

    initWorldKernel<<<1, 1>>>(
        d_world,
        d_triangles,
        triangleSize,
        d_spheres,
        sphereSize,
        d_lightIndices,
        lightIndexSize,
        d_materialLookup
    );
    checkCudaErrors(hipDeviceSynchronize());
}

void CUDAGlobals::copySceneData(const SceneData &sceneData)
{
    checkCudaErrors(hipMemcpy(
        d_triangles,
        sceneData.triangles.data(),
        sceneData.triangles.size() * sizeof(Triangle),
        hipMemcpyHostToDevice
    ));

    checkCudaErrors(hipMemcpy(
        d_spheres,
        sceneData.spheres.data(),
        sceneData.spheres.size() * sizeof(Sphere),
        hipMemcpyHostToDevice
    ));

    checkCudaErrors(hipMemcpy(
        d_lightIndices,
        sceneData.lightIndices.data(),
        sceneData.lightIndices.size() * sizeof(int),
        hipMemcpyHostToDevice
    ));
}

}
