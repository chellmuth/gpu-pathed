#include "hip/hip_runtime.h"
#include "scene.h"

#include "material.h"
#include "sphere.h"
#include "triangle.h"

namespace rays {

__device__ static Vec3 rotateY(Vec3 vector, float theta)
{
    return Vec3(
        cos(theta) * vector.x() - sin(theta) * vector.z(),
        vector.y(),
        sin(theta) * vector.x() + cos(theta) * vector.z()
    );
}

__global__ void createWorld(
    Primitive **primitives,
    PrimitiveList **world,
    Vec3 color,
    float lightPosition,
    bool update
) {
    if (update) {
        for (int i = 0; i < primitiveCount; i++) {
            delete(primitives[i]);
        }
        delete(*world);
    }

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int i = 0;
        primitives[i++] = new Sphere(
            Vec3(0.f, 0.f, -1.f),
            0.8f,
            new Material(color)
        );

        const float theta = lightPosition * M_PI;
        primitives[i++] = new Triangle(
            rotateY(Vec3(-0.5f, 0.6f, -2.f), theta),
            rotateY(Vec3(0.3f, 0.6f, -2.f), theta),
            rotateY(Vec3(-0.5f, 1.2f, -1.8f), theta),
            new Material(Vec3(1.f, 0.2f, 1.f), Vec3(14.f, 14.f, 14.f))
        );
        primitives[i++] = new Triangle(
            rotateY(Vec3(0.3f, 0.6f, -2.f), theta),
            rotateY(Vec3(0.3f, 1.2f, -1.8f), theta),
            rotateY(Vec3(-0.5f, 1.2f, -1.8f), theta),
            new Material(Vec3(1.f, 0.2f, 1.f), Vec3(14.f, 14.f, 14.f))
        );

        primitives[i++] = new Sphere(
            Vec3(0.f, -100.8f, -1.f),
            100.f,
            new Material(Vec3(1.f, 1.f, 1.f))
        );

        *world = new PrimitiveList(primitives, i);
    }
}

}
