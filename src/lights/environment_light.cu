#include "lights/environment_light.h"

#include <iostream>

#include "tinyexr.h"

#include "math/distribution.h"
#include "macro_helper.h"

#define checkCUDA(result) { gpuAssert((result), __FILE__, __LINE__); }

namespace rays {

EnvironmentLight EnvironmentLightParams::createEnvironmentLight() const {
    float *data;
    int width, height;

    const char *error = nullptr;
    const int code = LoadEXR(&data, &width, &height, m_filename.c_str(), &error);
    if (code == TINYEXR_SUCCESS) {
        std::cout << "Loaded environment light { "
                  << " width: " << width
                  << " height: " << height
                  << " }" << std::endl;
    } else {
        fprintf(stderr, "ENVIRONMENT MAP ERROR: %s\n", error);
        FreeEXRErrorMessage(error);
    }

    float *d_data;
    const size_t dataSize = width * height * 4 * sizeof(float);
    checkCUDA(hipMalloc((void **)&d_data, dataSize));
    checkCUDA(hipMemcpy(
        d_data,
        data,
        dataSize,
        hipMemcpyHostToDevice
    ));


    std::vector<float> intensities(width * height);
    for (int i = 0; i < width * height; i++) {
        intensities[i] = data[i * 4 + 0] + data[i * 4 + 1] + data[i * 4 + 2];
    }

    PhiThetaDistributionBuilder distributionBuilder(intensities.data(), width, height);
    auto distribution = distributionBuilder.buildPhiThetaDistribution();

    free(data);

    const EnvironmentLight environmentLight(d_data, distribution, width, height);
    return environmentLight;
}

}
